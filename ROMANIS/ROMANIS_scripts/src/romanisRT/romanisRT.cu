#include "hip/hip_runtime.h"
/*##################################################################################################*/
/*##################################################################################################

THIS IS THE AIO REAL-TIME VERSION.

/*##################################################################################################*/
/*##################################################################################################*/

// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <string.h>
#include <sys/types.h>
#include <sys/socket.h>
#include <sys/un.h>
#include <unistd.h>
#include <pthread.h>
#include <math.h>

// includes, cuda
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions
// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <hip/hip_vector_types.h>

#include "romanisRT.h"
using namespace std;


#define MAX2(x,y)      ((x)<(y) ? (y) : (x))

#define NUM_DAQS 8

//Macros for CUDA 5.5 compatibility
#define CUT_SAFE_CALL(x) checkCudaErrors((hipError_t)x)
#define cutGetMaxGflopsDeviceId() gpuGetMaxGflopsDeviceId()

StopWatchInterface *timer = NULL;

float hDFTCos[FBINS * DFTSIZE];
float hDFTSin[FBINS * DFTSIZE];
float hScale[SENSORS];

__constant__ float dDFTCos[FBINS * DFTSIZE];
__constant__ float dDFTSin[FBINS * DFTSIZE];
__constant__ float dScale[SENSORS];

float hBFRe[FBINS * BEAMS * SENSORS];
float hBFIm[FBINS * BEAMS * SENSORS];

float* dBFRe;
float* dBFIm;

short hSensorData [SENSORS * SAMPLES];
float hBFData[2*FBINS * BEAMS * BLOCKS];

float hostBFO[FBINS * BEAMS * BLOCKS];

short* dSensorData;
float* dBFData;
float *devBFO;
float *dbfoData;

//Struct for pre-recorded beam processing thread parameters
struct beamRecordedArgs {
    char *dataPath;
    char *bfOutputFile;
    };

float bfoData[FRAMES_PER_BLOCK*BEAMS*FBINS];

float *curDispBuf;
int dispBufPtr;
/*========================================================================================*/
/* VARIABLES FOR ROMANIS DAQ SERVER  */
struct DatStruct RomanisDat[NUM_OF_DAQS][N_BUFFERS];

int daqCount[NUM_OF_DAQS];
int daqStatus[NUM_OF_DAQS];
int daqWriteStart[NUM_OF_DAQS];
int startBF = WAIT;

pthread_mutex_t stsMutex = PTHREAD_MUTEX_INITIALIZER;
pthread_cond_t  stsCond  = PTHREAD_COND_INITIALIZER;
pthread_mutex_t daqMutex = PTHREAD_MUTEX_INITIALIZER;

void error(char *msg) {
    perror(msg);
    exit(1);
    }


/*************************************/

int bfDataCount = 0, totalBF = 0;

//Buffer pointer to store the display-processed beam frames
float frameBuffer[NUM_SQRS];

// Common variables used for data processing and display
int beams,curFrame,dataIdx,fRange,bfoBufPtr,drawBufPtr;
int frameCount = 0;
float cimg[ybeams][xbeams];

float dispBufBank[DISPLAY_BUFS][FRAMES_PER_BLOCK*BEAMS*FBINS];
int disBufStatus[DISPLAY_BUFS];
int disBufFrames[DISPLAY_BUFS];

/*************************************/

/*------------------------------------------------------------------------------------------------*/
/*------------------------------------------------------------------------------------------------*/
/*------------------------------------------------------------------------------------------------*/

/******** readDAQ() *********************
 There is a separate instance of this function
 for each connection.  It handles all communication
 once a connnection has been established.
 *****************************************/
void* readDAQ (void* pointer) {

    struct threadStruct * threadArgs = (struct threadStruct *) pointer;

    int sock = threadArgs->socket;
    int daqRead = 0;
    char *filename;
    short int i;

    int res,wr;

    struct DatStruct *ptr_0;
    void *buf;
    char start_code=START_CODE,err=0;
    struct sockaddr_in client_addr;
    socklen_t client_len;


    client_len = sizeof(client_addr);
    getpeername(sock, (struct sockaddr*)&client_addr, &client_len);

    switch(client_addr.sin_addr.s_addr & 0xFFFF0000) {
        case 0x0a000000 :
            filename = "DAQ_1";
            daqRead = DAQ1;
            break;
        case 0x0b000000 :
            filename = "DAQ_2";
            daqRead = DAQ2;
            break;
        case 0x0c000000 :
            filename = "DAQ_3";
            daqRead = DAQ3;
            break;
        case 0x0d000000 :
            filename = "DAQ_4";
            daqRead = DAQ4;
            break;

        case 0x0a0a0000 :
            filename = "DAQ_5";
            daqRead = DAQ5;
            break;
        case 0x0b0a0000 :
            filename = "DAQ_6";
            daqRead = DAQ6;
            break;
        case 0x0c0a0000 :
            filename = "DAQ_7";
            daqRead = DAQ7;
            break;
        case 0x0d0a0000 :
            filename = "DAQ_8";
            daqRead = DAQ8;
            break;
        }


    ptr_0 = &RomanisDat[daqRead][0];

    wr = write(sock, &start_code, sizeof(char));

    if(wr<0) {
        printf("Failed to write the start code for %s\n",filename);
        pthread_exit(0);
        }
    daqStatus[daqRead] = READ_IN_PROGRESS;

    while(!(ptr_0->Done)) {

        ptr_0->Status = READ_IN_PROGRESS;

        buf = (void *)&(ptr_0->Buffer);

        while(1) {
            res = read(sock, (buf + ptr_0->DataLen), (BUFFER_SIZE - ptr_0->DataLen));
            if (0 > res) {
                printf("%s: Error Reading from Socket [res= %d]\n", filename, res);
                ptr_0->err = 1;
                goto OUTLOOP;
                }

            if (0 == res) {		//No more data from source
                ptr_0->Done = 1;
                break;
                }

            ptr_0->DataLen += res;

            if (ptr_0->DataLen == BUFFER_SIZE)
                break;
            }


        ptr_0->Status = READ_COMPLETED;
        if(ptr_0->bfStatus == BF_COMPLETED) {
            ptr_0->bfStatus = BF_READY;
            }
        else {

            printf("DAQ BF STATUS CORRUPT  current status : %x , expected status : %x , wanted to change to : %x !!!!\n",ptr_0->bfStatus,BF_COMPLETED,BF_READY );
            }

        daqCount[daqRead]++;

        if(startBF == WAIT) {
            pthread_mutex_lock( &stsMutex );

            if((daqCount[0]*daqCount[1]*daqCount[2]*daqCount[3]*daqCount[4]*daqCount[5]*daqCount[6]*daqCount[7])) {
                startBF = START;
                pthread_cond_signal( &stsCond );
                printf("All DAQs have values ... Signalling beamformer to start processing ... \n");
                }
            pthread_mutex_unlock( &stsMutex );
            }

        if(!ptr_0->Done) {
            ptr_0 = ptr_0->next;
            }
        }



OUTLOOP:

    for(i=0; i<N_BUFFERS; i++) {
        if(RomanisDat[daqRead][i].err) {
            err = 1 ;
            break;
            }
        }

    if(!err)
        printf("%s: Transfer Complete\n",filename);
    daqStatus[daqRead] = READ_COMPLETED;
    pthread_exit(0);
    }

//################################################################################################//

void * daqWrite (void *args) {

    int daqNum = *((int *)args);
    char *filename;
    void *buf_1;
    int snd;
    char *f_path, *base_dir = DATA_FOLDER;
    FILE *fp;
    int totalRead = 0;

    switch(daqNum) {
        case DAQ1 :
            filename = "DAQ_1";
            break;
        case DAQ2 :
            filename = "DAQ_2";
            break;
        case DAQ3 :
            filename = "DAQ_3";
            break;
        case DAQ4 :
            filename = "DAQ_4";
            break;

        case DAQ5 :
            filename = "DAQ_5";
            break;
        case DAQ6 :
            filename = "DAQ_6";
            break;
        case DAQ7 :
            filename = "DAQ_7";
            break;
        case DAQ8 :
            filename = "DAQ_8";
            break;

        default  :
            filename = "DAQ_n";
            break;
        }

    printf("Starting write for %s ...\n",filename );
    f_path = (char *)malloc((strlen(base_dir) + strlen(filename) + 1)*sizeof(char));
    strcpy(f_path, base_dir);
    strcat(f_path, filename);

    fp = fopen(f_path,"w+");
    struct DatStruct *ptr_0;
    ptr_0 = &RomanisDat[daqNum][0];
    int wc = 0,dataLen=0,transfer_bytes=0;
    while(totalRead!=daqCount[daqNum] || daqStatus[daqNum]!= READ_COMPLETED) {

        buf_1 = (void *)&(ptr_0->Buffer);
        wc=0;
        if(ptr_0->Status == READ_COMPLETED) {

            dataLen = ptr_0->DataLen;
            while(dataLen) {
                transfer_bytes = (dataLen > WRITE_BLOCK) ? WRITE_BLOCK : dataLen;
                snd = (int)fwrite((buf_1 + wc),1,transfer_bytes,fp);

                if (snd < 0) {
                    printf("%s: Error Writing to Disk [snd= %d]\n", filename, snd);
                    fclose(fp);
                    free(filename);
                    pthread_exit(0);
                    }
                dataLen -= snd;
                wc += snd;
                }

            totalRead++;
            ptr_0->Status = WRITE_COMPLETED;
            ptr_0 = ptr_0->next;

            }
        else {
            usleep(500);
            }
        }
    fclose(fp);

    printf("File write completed for %s .. \n",filename );
    pthread_exit(0);
    }


//################################################################################################//
/*------------------------------------------------------------------------------------------------*/
/*------------------------------------------------------------------------------------------------*/
/*------------------------------------------------------------------------------------------------*/


/* initCuda() : Select the GPU to be used and initialize it  */
void initCuda(void) {

    CUT_SAFE_CALL(hipDeviceReset());
    int dev = cutGetMaxGflopsDeviceId();
    printf("MAX Device ID : %d\n", dev);
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, dev);
    printf("DEIVE NAME : %s\n", properties.name);
    CUT_SAFE_CALL(hipSetDevice(dev));
    CUT_SAFE_CALL(gpuDeviceInit(dev));
    }

/* checkCudaMem() : Check for available memory in the GPU   */
void checkCudaMem(void) {
    size_t total, avail;
    CUT_SAFE_CALL(hipMemGetInfo(&avail, &total));
    printf("CUDA Memory: total = %ld, free = %ld\n", total, avail);
    }

/* allocCudaMem() : Allocate memory in the GPU */
void allocCudaMem(void) {
    CUT_SAFE_CALL(hipMalloc((void**)&dSensorData, sizeof(short)*SENSORS*DFTSIZE*BLOCKS));
    CUT_SAFE_CALL(hipMalloc((void** )&dBFData, sizeof(hBFData)));
    CUT_SAFE_CALL(hipMalloc((void** )&dBFRe, sizeof(hBFRe)));
    CUT_SAFE_CALL(hipMalloc((void** )&dBFIm, sizeof(hBFIm)));
    CUT_SAFE_CALL(hipMalloc((void** )&devBFO, sizeof(hostBFO)));
    CUT_SAFE_CALL(hipMalloc((void** )&dbfoData, sizeof(bfoData)));
    }

/* shutdown() : Free up GPU memory and reset GPU  */
void shutdown(void) {
    CUT_SAFE_CALL(hipFree(dBFRe));
    CUT_SAFE_CALL(hipFree(dBFIm));
    CUT_SAFE_CALL(hipFree(dBFData));
    CUT_SAFE_CALL(hipFree(dSensorData));
    CUT_SAFE_CALL(hipFree(devBFO));
    CUT_SAFE_CALL(hipFree(dbfoData));
    CUT_SAFE_CALL(hipDeviceReset());
    }

/* initDFT() : Initialize the DFT arrays  */
void initDFT(void) {
    float K = 2 * PI / DFTSIZE;
    for (int i = 0; i < DFTSIZE; i++) {
        for (int f = 0; f < FBINS; f++) {
            hDFTCos[f * DFTSIZE + i] = cos(K * (f + FBIN0) * i);
            hDFTSin[f * DFTSIZE + i] = -sin(K * (f + FBIN0) * i);
            }
        }
    CUT_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dDFTSin), hDFTSin, sizeof(hDFTSin)));
    CUT_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dDFTCos), hDFTCos, sizeof(hDFTCos)));
    }

/* initBF() : Initialize the beamformer arrays  */
void initBF() {
    for (int e = 0; e < EBEAMS; e++) {
        float phi = e * BEAMSPC - (EBEAMS - 1) * BEAMSPC / 2;
        for (int a = 0; a < ABEAMS; a++) {
            float theta = a * BEAMSPC - (ABEAMS - 1) * BEAMSPC / 2;
            for (int f = 0; f < FBINS; f++) {
                int fb = FBIN0 + f;
                for (int i = 0; i < SENSORS; i++) {
                    float w = sensorXPos[i] * sin(theta) * cos(phi)
                              + sensorYPos[i] * sin(phi);
                    w *= 2 * PI * RATE / DFTSIZE * fb / SOUNDSPEED;
                    int tn = (f * SENSORS + i) * BEAMS + e * ABEAMS + a;
                    hBFRe[tn] = sin(w) * hScale[i];
                    hBFIm[tn] = cos(w) * hScale[i];
                    }
                }
            }
        }
    CUT_SAFE_CALL(hipMemcpy(dBFRe, hBFRe, sizeof(hBFRe), hipMemcpyHostToDevice));
    CUT_SAFE_CALL(hipMemcpy(dBFIm, hBFIm, sizeof(hBFIm), hipMemcpyHostToDevice));
    }

/* readScale() : Read calibration of sensors  */
void readScale() {

    FILE* fh = fopen("calib.txt", "rt");
    if (fh == NULL) {
        printf("Sensor calibration not available\n");
        for (int i = 0; i < SENSORS; i++)
            hScale[i] = 1;
        }
    else {
        for (int i = 0; i < SENSORS; i++) {
            float s, o;
            fscanf(fh, "%f %f", &s, &o);
            hScale[i] = s;
            }
        fclose(fh);
        }
    CUT_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dScale), hScale, sizeof(hScale)));
    }


/* process() : beam-former CUDA kernel  */

__global__ void process(short* dataIn, float* dataOut, float* dBFRe,float* dBFIm,float *devBFO) {
    __shared__ float re[SENSORS];
    __shared__ float im[SENSORS];
    int f = blockIdx.x;
    int n = blockIdx.y;
    int s = threadIdx.x;

    int offSet1,offSet2;

    if (s < SENSORS) {
        float sumRe = 0;
        float sumIm = 0;
        offSet1 = f * DFTSIZE;
        offSet2 = n * DFTSIZE*SENSORS;

        for (int i = 0; i < DFTSIZE; i++) {
            sumRe += dDFTCos[offSet1 + i]* dataIn[offSet2 + s];
            sumIm += dDFTSin[offSet1 + i]* dataIn[offSet2 + s];
            offSet2+=SENSORS;
            }
        re[s] = sumRe;
        im[s] = sumIm;
        }
    __syncthreads();
    if (s < BEAMBLKS * BEAMS) {
        int b = s % BEAMS;
        int start = s / BEAMS * SENSORS / BEAMBLKS;
        float sumRe = 0;
        float sumIm = 0;
        offSet1 = f * SENSORS* BEAMS;
        for (int i = start; i < start + SENSORS / BEAMBLKS; i++) {
            int tn = offSet1 + b;
            float tr = re[i] * dBFRe[tn] - im[i] * dBFIm[tn];
            float ti = im[i] * dBFRe[tn] + re[i] * dBFIm[tn];
            sumRe += tr;
            sumIm += ti;
            offSet1+=BEAMS;
            }

        int outndx2 = 2 * ((n * BEAMS + b) * FBINS + f);
        if (s < BEAMS) {
            dataOut[outndx2] = sumRe;
            dataOut[outndx2 + 1] = sumIm;
            }

        __syncthreads();
        if (s >= BEAMS) {
            dataOut[outndx2] += sumRe;
            dataOut[outndx2 + 1] += sumIm;
            }

        }

    __syncthreads();
    int b = s % BEAMS;
    int outndx2 = 2 * ((n * BEAMS + b) * FBINS + f);
    //devBFO[outndx2/2] = sqrt(pow(dataOut[outndx2],2)+pow(dataOut[outndx2 + 1],2));
    devBFO[outndx2/2] = (fabs(dataOut[outndx2])+fabs(dataOut[outndx2 + 1]))/2;

    }

/* ###################################################################################*/
/* ###################################################################################*/
template <typename T>
std::string to_string(T value) {
    std::ostringstream os ;
    os << value ;
    return os.str() ;
    }


// Initializes all the parameters and variables
void initVars() {
    fRange = freqBinUpper-freqBinLower+1;
    beams = ybeams*xbeams*fbins;
    curFrame = 0;
    //intialize the cimg array to all zeros
    for(int y=0; y<ybeams; y++) {
        for(int x=0; x<xbeams; x++) {
            cimg[y][x]=0.0f;
            }
        }
    dataIdx = 0;

    for(int i=0; i<DISPLAY_BUFS; i++) {
        for(int j=0; j<FRAMES_PER_BLOCK*BEAMS*FBINS; j++) {
            dispBufBank[i][j] = 0.0;
            }
        disBufStatus[i] = HOLD;
        }

    //Initialization
    for(int d=0; d<NUM_OF_DAQS; d++) {
        for(int i=0; i<N_BUFFERS; i++) {
            RomanisDat[d][i].err = 0;
            RomanisDat[d][i].DataLen = 0;
            RomanisDat[d][i].Status = WRITE_COMPLETED;
            RomanisDat[d][i].bfStatus = BF_COMPLETED;
            RomanisDat[d][i].Done = 0;
            if((N_BUFFERS-1) == i)
                RomanisDat[d][i].next = (struct DatStruct*)&RomanisDat[d][0];
            else
                RomanisDat[d][i].next = (struct DatStruct*)&RomanisDat[d][i+1];
            }
        }

    dispBufPtr = 0;
    }
/* ###################################################################################*/
/* ###################################################################################*/
// Interpolation Function
float interpolate (float in[ybeams][xbeams], float out[ybeamsInt][xbeamsInt]) {

    float xStep,yStep;
    float maxVal = 0.0;
    /* Fill all first rows */
    for(int j=0; j<ybeams; j++) {
        for(int i = 0; i<xbeams-1; i++) {
            xStep = (in[j][i+1] - in[j][i])/(INTERPOLATION_FACTOR);
            out[INTERPOLATION_FACTOR*j][INTERPOLATION_FACTOR*i] = in[j][i];
            for(int k=1; k<INTERPOLATION_FACTOR; k++) {
                out[INTERPOLATION_FACTOR*j][INTERPOLATION_FACTOR*i+k] = in[j][i] + k*xStep;
                if (maxVal<out[INTERPOLATION_FACTOR*j][INTERPOLATION_FACTOR*i+k])
                    maxVal = out[INTERPOLATION_FACTOR*j][INTERPOLATION_FACTOR*i+k];
                }
            }
        out[INTERPOLATION_FACTOR*j][xbeamsInt-1] = in[j][xbeams-1];
        if (maxVal<out[INTERPOLATION_FACTOR*j][xbeamsInt-1])
            maxVal = out[INTERPOLATION_FACTOR*j][xbeamsInt-1];
        }

    /* Fill up all the clomuns */
    for(int i=0; i<ybeams-1; i++) {
        for(int j=0; j<xbeamsInt; j++) {
            yStep = (out[INTERPOLATION_FACTOR*(i+1)][j] - out[INTERPOLATION_FACTOR*i][j])/INTERPOLATION_FACTOR;
            for(int k=1; k<INTERPOLATION_FACTOR; k++) {
                out[INTERPOLATION_FACTOR*i+k][j] = out[INTERPOLATION_FACTOR*i][j] + k*yStep;
                if (maxVal<out[INTERPOLATION_FACTOR*i+k][j])
                    maxVal = out[INTERPOLATION_FACTOR*i+k][j];

                }
            }
        }

    return 20*log10(maxVal);
    }
/* ###################################################################################
                    Beamformer Output Processing
   ################################################################################### */

void preprocBF(float *hBFData,float *bfoData, int framesToDraw ) {


    float bfoFrameAvg[FrameSize][beams];
    float bfoSum = 0.0;

    int k;
    int dataCount = 0;

    for(k=0; k<framesToDraw; k++) {

        int i,j;
        for( i=0; i<beams; i++) {
            bfoSum = 0.0;
            for( j=0; j<FrameSize; j++) {
                bfoSum += hBFData[FrameSize*beams*k+beams*j+i];
                }
            bfoData[beams*k+i] = bfoSum/FrameSize;
            }

        }
    }

__global__ void preProcBF(float *dataIn, float *dataOut) {

    int dataPtr = threadIdx.x * FBINS*BEAMS*FRAMESIZE;
    int dataOutPtr = threadIdx.x * FBINS*BEAMS;
    float *threadBuf = dataIn + dataPtr;
    int i,j,dataCount = 0;;
    float bfoSum;

    for( i=0; i<FBINS*BEAMS; i++) {
            bfoSum = 0.0;
            dataCount = i;
            for( j=0; j<FRAMESIZE; j++) {
                bfoSum += threadBuf[dataCount];
                dataCount+=BEAMS*FBINS;
                }
            dataOut[dataOutPtr+i] = bfoSum/FRAMESIZE;
            }

    }

void processBF(float *bfOutput) {

    float sum,maxVal;
    float beamFrames[xbeams][ybeams][fbins];
    float frameFreqAvg[xbeams][ybeams];
    float cimgInt[ybeamsInt][xbeamsInt];
    int bufIdx = 0;
    int dynAdd;

    sum =0;
    maxVal = 0;

//Select the frames in blocks of Framesize,Average out the frames over the desired frequency range

    for(int y=ybeams-1; y>-1; y--) {
        for(int x=xbeams-1; x>-1; x--) {

            //dataIdx += FBINS*BEAMS*(freqBinLower-1);
            for(int f=0; f<fbins; f++) {
                beamFrames[x][y][f] = bfOutput[dataIdx];
                dataIdx++;
                }
            sum = 0.0f;
            for(int f=freqBinLower-1; f<freqBinUpper; f++) {
                sum += beamFrames[x][y][f];
                }
            frameFreqAvg[x][y] = sum/fRange;
            cimg[y][x] = ExpAvg*cimg[y][x] + (1.0-ExpAvg)*(sum/fRange);
            }
        }


//Interpolate cimg
    maxVal = interpolate(cimg,cimgInt);
    dynAdd = -maxVal+DynRange;

//Take log of interpolated cimg and do dynamic ranging
float temp;
    for(int y=0; y<ybeamsInt; y++) {
        for(int x=0; x<xbeamsInt; x++) {
            temp = (20*log10(cimgInt[y][x])+dynAdd)/DynRange;
            if(temp<0)
                frameBuffer[y*xbeamsInt + x +y-2] = 0.0;
            else if(temp>1.0)
                frameBuffer[y*xbeamsInt + x +y-2] = 63.0;
            else
                frameBuffer[y*xbeamsInt + x +y-2] = 63.0*temp;
            }
        }
    }

/******************BEAM PROCESSING FOR PRE-RECORDED DATA SET********************************************/
void * beamProcess(void *args) {

    printf("Starting beam-forming .... \n");

    int hinCount;

    int i=0,blks;
    int samples = 0;

    int rounds = 0;

    int offSet = 0;

    readScale();

    initCuda();
    checkCudaMem();
    initSensors();
    allocCudaMem();
    checkCudaMem();
    initDFT();
    initBF();

    struct DatStruct *ptr_daq1 = &RomanisDat[DAQ1][0];
    struct DatStruct *ptr_daq2 = &RomanisDat[DAQ2][0];
    struct DatStruct *ptr_daq3 = &RomanisDat[DAQ3][0];
    struct DatStruct *ptr_daq4 = &RomanisDat[DAQ4][0];
    struct DatStruct *ptr_daq5 = &RomanisDat[DAQ5][0];
    struct DatStruct *ptr_daq6 = &RomanisDat[DAQ6][0];
    struct DatStruct *ptr_daq7 = &RomanisDat[DAQ7][0];
    struct DatStruct *ptr_daq8 = &RomanisDat[DAQ8][0];

    bfDataCount = 0;
    totalBF = 0;
    printf("Beam-former Initialized ... Start DAQ accquisition ... \n");

    pthread_mutex_lock( &stsMutex );
    while( startBF != START ) {
        pthread_cond_wait( &stsCond, &stsMutex );
        }
    pthread_mutex_unlock( &stsMutex );

    printf("Starting beam-forming process ..... \n");
    while(1) {
        samples = 0;

        if(ptr_daq1->bfStatus == BF_READY && ptr_daq3->bfStatus == BF_READY && ptr_daq5->bfStatus == BF_READY && ptr_daq7->bfStatus == BF_READY &&
                ptr_daq2->bfStatus == BF_READY && ptr_daq4->bfStatus == BF_READY && ptr_daq6->bfStatus == BF_READY && ptr_daq8->bfStatus == BF_READY ) {


            for(int daq=0; daq<NUM_OF_DAQS; daq++) {

                int n = (RomanisDat[daq][totalBF].DataLen - offSet)/( (64 * sizeof(short) + 4));
                if(daq==0)
                    samples = n;
                else if (n < samples)
                    samples = n;
                }

            int daqPtr = offSet/2 ,bufPtr = 0;

            short *curBuf;

            for (int j = 0; j < 0.75*samples; j++) {

                daqPtr+=2;

                curBuf = ptr_daq1->Buffer + offSet;
                memcpy(hSensorData+bufPtr,curBuf+daqPtr,64*sizeof(short));
                bufPtr+=64;

                curBuf = ptr_daq2->Buffer + offSet;
                memcpy(hSensorData+bufPtr,curBuf+daqPtr,64*sizeof(short));
                bufPtr+=64;

                curBuf = ptr_daq3->Buffer + offSet;
                memcpy(hSensorData+bufPtr,curBuf+daqPtr,64*sizeof(short));
                bufPtr+=64;

                curBuf = ptr_daq4->Buffer + offSet;
                memcpy(hSensorData+bufPtr,curBuf+daqPtr,64*sizeof(short));
                bufPtr+=64;

                curBuf = ptr_daq5->Buffer + offSet;
                memcpy(hSensorData+bufPtr,curBuf+daqPtr,64*sizeof(short));
                bufPtr+=64;

                curBuf = ptr_daq6->Buffer + offSet;
                memcpy(hSensorData+bufPtr,curBuf+daqPtr,64*sizeof(short));
                bufPtr+=64;

                curBuf = ptr_daq7->Buffer + offSet;
                memcpy(hSensorData+bufPtr,curBuf+daqPtr,64*sizeof(short));
                bufPtr+=64;

                curBuf = ptr_daq8->Buffer + offSet;
                memcpy(hSensorData+bufPtr,curBuf+daqPtr,60*sizeof(short));
                bufPtr+=60;

                daqPtr+=64;

				if(j%4 == 0) daqPtr+=66;

                }

            offSet = 132 - ((RomanisDat[0][totalBF].DataLen - offSet) % (64 * sizeof(short) + 4));


            i = 0;

            while (i < samples) {
                blks = (samples - i) / DFTSIZE;
                if (blks > BLOCKS)
                    blks = BLOCKS;

                CUT_SAFE_CALL(hipMemcpy(dSensorData, hSensorData+SENSORS*i, DFTSIZE*SENSORS*blks*sizeof(short), hipMemcpyHostToDevice));
                dim3 threads(MAX2(SENSORS,BEAMS*BEAMBLKS));
                dim3 grid(FBINS, blks);
                process<<<grid, threads>>>(dSensorData, dBFData, dBFRe, dBFIm,devBFO);


                dim3 threads1(blks/FRAMESIZE);
                dim3 grid1(1, 1);
                preProcBF<<<grid1, threads1>>>(devBFO, dbfoData);
                CUT_SAFE_CALL(hipMemcpy(bfoData, dbfoData, sizeof(bfoData),hipMemcpyDeviceToHost));

                for(int j=0; j<(blks/FRAMESIZE)*BEAMS*FBINS; j++)
                    dispBufBank[bfDataCount%DISPLAY_BUFS][j] = bfoData[j];
                disBufFrames[bfDataCount%DISPLAY_BUFS] = blks/FRAMESIZE;
                disBufStatus[bfDataCount%DISPLAY_BUFS] = DRAW;
                bfDataCount++;


                i += DFTSIZE * BLOCKS;

                }

            totalBF++;
            int nextBuf = totalBF%N_BUFFERS;

            ptr_daq1->bfStatus = BF_COMPLETED;
            ptr_daq1 = &RomanisDat[DAQ1][nextBuf];
            ptr_daq3->bfStatus = BF_COMPLETED;
            ptr_daq3 = &RomanisDat[DAQ2][nextBuf];
            ptr_daq5->bfStatus = BF_COMPLETED;
            ptr_daq5 = &RomanisDat[DAQ3][nextBuf];
            ptr_daq7->bfStatus = BF_COMPLETED;
            ptr_daq7 = &RomanisDat[DAQ4][nextBuf];
            ptr_daq2->bfStatus = BF_COMPLETED;
            ptr_daq2 = &RomanisDat[DAQ5][nextBuf];
            ptr_daq4->bfStatus = BF_COMPLETED;
            ptr_daq4 = &RomanisDat[DAQ6][nextBuf];
            ptr_daq6->bfStatus = BF_COMPLETED;
            ptr_daq6 = &RomanisDat[DAQ7][nextBuf];
            ptr_daq8->bfStatus = BF_COMPLETED;
            ptr_daq8 = &RomanisDat[DAQ8][nextBuf];


            }
        //Exit condition
        else if(totalBF==daqCount[DAQ1] && daqStatus[DAQ1]== READ_COMPLETED) {
            printf("BF completed ... \n");
            break;
            }
        else {
            usleep(500);
            }
        }

    shutdown();
    printf("Total data blocks processed in BF : %d\n",totalBF);
    printf("Total blocks sent to display : %d\n",bfDataCount);
    pthread_exit(0);
    }
/**********************************************************************************************/

/* ###################################################################################
					Beamformer Display using OpenGL
   ################################################################################### */

void printText(int x, int y, string textDisplay) {

    glMatrixMode(GL_PROJECTION);
    glPushMatrix();
    glLoadIdentity();
    glOrtho(0, WIN_WIDTH, 0, WIN_HEIGHT, -1.0f, 1.0f);
    glMatrixMode(GL_MODELVIEW);
    glPushMatrix();
    glLoadIdentity();
    glPushAttrib(GL_DEPTH_TEST);
    glDisable(GL_DEPTH_TEST);
    glRasterPos2i(x,y);
    for (int i=0; i<textDisplay.size(); i++) {
        glutBitmapCharacter(GLUT_BITMAP_HELVETICA_12, textDisplay[i]);
        }
    glPopAttrib();
    glMatrixMode(GL_PROJECTION);
    glPopMatrix();
    glMatrixMode(GL_MODELVIEW);
    glPopMatrix();
    }

void displayMsg() {

    //Set Colour to white
    glColor3f(1.0,1.0,1.0);
    string textDisplay;

    //Display all option parameters
    textDisplay="Current Lower Frequency Bin : "+to_string(freqBinLower);
    printText(10,7,textDisplay);
    textDisplay="Current Lower Frequency Bin : "+to_string(freqBinUpper);
    printText(10,24,textDisplay);
    textDisplay="Current Dynamic Range Limit : "+to_string(DynRange);
    printText(10,41,textDisplay);

    //Display guide to change values of the parameters
    textDisplay="Press UP key to raise and  DOWN key to lower the Lower Frequency Bin limit";
    printText(270,41,textDisplay);
    textDisplay="Press RIGHT key to raise and LEFT key to lower the Upper Frequency Bin limit";
    printText(270,24,textDisplay);

    //Display Cloour bar text
    textDisplay="Colour Bar for Signal Strength  : ";
    printText(10,115,textDisplay);

    //Display color bar values
    textDisplay="  0               10               20               30               40                50               60               70                 80 ";
    printText(30,80,textDisplay);

    //Display azimuthal angle values
    textDisplay="-8.8 -7.2  -6.4  -5.6  -4.8  -4.0  -3.2  -2.4  -1.6  -0.8   0.0   0.8   1.6   2.4   3.2    4.0   4.8   5.6   6.4   7.2   8.0   8.8";
    printText(25,165,textDisplay);
    textDisplay="Azimuthal";
    printText(320,150,textDisplay);

    //Display Elevation angle values
    textDisplay="-4.0";
    printText(10,180,textDisplay);
    textDisplay="-3.2";
    printText(10,213.5,textDisplay);
    textDisplay="-2.4";
    printText(10,247,textDisplay);
    textDisplay="-1.6";
    printText(10,280.5,textDisplay);
    textDisplay="-0.8";
    printText(10,314,textDisplay);
    textDisplay="0.0";
    printText(15,347.5,textDisplay);
    textDisplay="0.8";
    printText(15,381,textDisplay);
    textDisplay="1.6";
    printText(15,414.5,textDisplay);
    textDisplay="2.4";
    printText(15,448,textDisplay);
    textDisplay="3.2";
    printText(15,481.5,textDisplay);
    textDisplay="4.0";
    printText(15,515,textDisplay);

    //Display "Elevation" as text
    textDisplay="E";
    printText(700,420,textDisplay);
    textDisplay="l";
    printText(700,405,textDisplay);
    textDisplay="e";
    printText(700,390,textDisplay);
    textDisplay="v";
    printText(700,375,textDisplay);
    textDisplay="a";
    printText(700,360,textDisplay);
    textDisplay="t";
    printText(700,345,textDisplay);
    textDisplay="i";
    printText(700,330,textDisplay);
    textDisplay="o";
    printText(700,315,textDisplay);
    textDisplay="n";
    printText(700,300,textDisplay);


    }
//! Create a timer event for refresh
void timerEvent(int value) {
    if(disBufStatus[dispBufPtr] == DRAW) {
        updateFrame();
        glutPostRedisplay();
        }
    glutTimerFunc(refreshDelay, timerEvent, 0);
    }

void genVerticesDisplay() {

    int i;
    int x=0;
    int y=0;
    int clr;
    float xcords[xPoints];
    float ycords[yPoints];

    int clrPtr;
    float xstep = (2*xLim)/(float)(xPoints-1);
    float ystep = (2*yLim)/(float)(yPoints-1);

    for(i=0; i<xPoints; i++)
        xcords[i] = -xLim + i*xstep;

    for(i=0; i<yPoints; i++)
        ycords[i] = -yLim + i*ystep+yOffset;

    clrPtr = 0;

    for (i=0; i<NUM_SQRS; i++) {
        if (x==xPoints-1) {
            x=0;
            y=y+1;
            }

        clr = (int)(frameBuffer[clrPtr]/2.0);
        if (clr < MAX_COLOURS)
            glColor3f(accousColor[clr][0],accousColor[clr][1],accousColor[clr][2]);
        else
            glColor3f(accousColor[MAX_COLOURS-1][0],accousColor[MAX_COLOURS-1][1],accousColor[MAX_COLOURS-1][2]);

        glVertex3f(xcords[x], ycords[y], 0.0);
        glVertex3f(xcords[x+1], ycords[y], 0.0);
        glVertex3f(xcords[x+1],  ycords[y+1], 0.0);
        glVertex3f( xcords[x], ycords[y+1], 0.0);

        x+=1;
        clrPtr+=1;

        }

    //Draw the colour bar
    x=0;
    float yColorBar[] = {-0.65,-0.6};
    int curColour = 0;
    int drawCnt = 0;

    for(i=0; i<4*xPoints; i+=4) {

        glColor3f(accousColor[curColour][0],accousColor[curColour][1],accousColor[curColour][2]);
        glVertex3f(xcords[x], yColorBar[0], 0.0);
        glVertex3f(xcords[x+1], yColorBar[0], 0.0);
        glVertex3f(xcords[x+1],  yColorBar[1], 0.0);
        glVertex3f( xcords[x], yColorBar[1], 0.0);

        drawCnt+=1;
        if(drawCnt % 5 == 0)
            curColour++;

        x+=1;
        }


    }

void fpsDisplay() {
    char fps[64];
    curTime = (float)frameCount*timePerFrame;
    sprintf(fps, "ROMANIS RT DISPLAY  Current Time : %0.3f  Frame Number : %d",curTime,frameCount);
    glutSetWindowTitle(fps);
    }

void display(void) {

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glEnable(GL_DEPTH_TEST);
    glClear( GL_COLOR_BUFFER_BIT);

    glShadeModel(GL_SMOOTH);
    glBegin(GL_QUADS);

    genVerticesDisplay();

    glEnd();
    glFlush();

    displayMsg();

    fpsDisplay();

    frameCount++;
    curFrame++;
    glutSwapBuffers();



    }

void special(int key, int x, int y) {
    switch (key) {
        case GLUT_KEY_LEFT:
            if(freqBinUpper > freqBinLower)
                freqBinUpper--;
            break;
        case GLUT_KEY_RIGHT:
            if(freqBinUpper<FBINS-1)
                freqBinUpper++;
            break;
        case GLUT_KEY_UP:
            if(freqBinLower<freqBinUpper)
                freqBinLower++;
            break;
        case GLUT_KEY_DOWN:
            if(freqBinLower>0)
                freqBinLower--;
            break;
        }

    glutPostRedisplay();
    }

void drawFrame(int argc, char *argv[]) {

    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
    glutInitWindowSize(WIN_WIDTH, WIN_HEIGHT);
    glutInitWindowPosition(0, 0);

    glutCreateWindow("ROMANIS BEAM VISULISATION ");

    glutTimerFunc(refreshDelay, timerEvent, 0);
    glutDisplayFunc(display);
    glutSpecialFunc(special);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glutMainLoop();
    }


void * startDraw(void *args) {

    char fakeParam[] = "fake";
    char *fakeargv[] = { fakeParam, NULL };
    int fakeargc = 1;
    initVars();
    drawFrame(fakeargc,fakeargv);

    pthread_exit(0);
    }

void updateFrame() {


    if(curFrame == disBufFrames[dispBufPtr] && frameCount!=0) {
        dataIdx = 0;
        curFrame = 0;
        memset(&dispBufBank[dispBufPtr][0],0.0,(FRAMES_PER_BLOCK*BEAMS*FBINS)*sizeof(float));
        disBufStatus[dispBufPtr] = HOLD;
        dispBufPtr++;
        if(dispBufPtr == DISPLAY_BUFS ) dispBufPtr = 0;

        }

    curDispBuf = &dispBufBank[dispBufPtr][0];

    processBF(curDispBuf);

    }
/* ###################################################################################
					 Main function
   ################################################################################### */
int main(int argc, char* argv[]) {

    initVars();
    int sockfd, newsockfd[NUM_OF_DAQS], portno;
    struct sockaddr_in serv_addr, cli_addr;
    socklen_t clilen;

    if (argc < 2) {
        fprintf(stderr,"ERROR, no port provided\n");
        exit(1);
        }
    sockfd = socket(AF_INET, SOCK_STREAM, 0);
    if (sockfd < 0)
        error("ERROR opening socket");
    bzero((char *) &serv_addr, sizeof(serv_addr));
    portno = atoi(argv[1]);
    serv_addr.sin_family = AF_INET;
    serv_addr.sin_addr.s_addr = INADDR_ANY;
    serv_addr.sin_port = htons(portno);
    if (bind(sockfd, (struct sockaddr *) &serv_addr,
             sizeof(serv_addr)) < 0)
        error("ERROR on binding");
    listen(sockfd,5);
    clilen = sizeof(cli_addr);
    signal(SIGCHLD,SIG_IGN);

    printf("\n");
    system("date;");

    int i;
    for(i=0; i<NUM_OF_DAQS; i++) {
        daqCount[i] = 0;
        daqWriteStart[i] = i;
        }

    pthread_t threadDAQ[NUM_OF_DAQS],threadWrite[NUM_OF_DAQS],threadBF;

    struct threadStruct daq[NUM_OF_DAQS];
    int daqNum = 0;

    printf("Launching BF thread now ... \n");
    pthread_create(&threadBF,NULL,beamProcess,(void *)&startBF);
    

    pthread_attr_t attrDAQ;
    pthread_attr_init(&attrDAQ);
    pthread_attr_setscope(&attrDAQ,PTHREAD_SCOPE_SYSTEM);

    while (daqNum < NUM_OF_DAQS) {
        newsockfd[daqNum] = accept(sockfd, (struct sockaddr *) &cli_addr, &clilen);

        if (newsockfd[daqNum] < 0)
            error("ERROR on accept");
        else {

            daq[daqNum].socket = newsockfd[daqNum];
            daq[daqNum].daqNum = daqNum;

            pthread_create(&threadWrite[daqNum],NULL,daqWrite,(void *)&daqWriteStart[daqNum]);
            pthread_create(&threadDAQ[daqNum],&attrDAQ,readDAQ,(void *)&daq[daqNum]);

            daqNum++;
            }
        } /* end of while */
        
    pthread_t threadDraw;
    void *dummy = 0x0000;
    pthread_create(&threadDraw,NULL,startDraw,dummy);
        
    pthread_attr_destroy(&attrDAQ);

    pthread_join(threadDAQ[0],NULL);
    pthread_join(threadDAQ[1],NULL);
    pthread_join(threadDAQ[2],NULL);
    pthread_join(threadDAQ[3],NULL);
    pthread_join(threadDAQ[4],NULL);
    pthread_join(threadDAQ[5],NULL);
    pthread_join(threadDAQ[6],NULL);
    pthread_join(threadDAQ[7],NULL);


    pthread_join(threadWrite[0],NULL);
    pthread_join(threadWrite[1],NULL);
    pthread_join(threadWrite[2],NULL);
    pthread_join(threadWrite[3],NULL);
    pthread_join(threadWrite[4],NULL);
    pthread_join(threadWrite[5],NULL);
    pthread_join(threadWrite[6],NULL);
    pthread_join(threadWrite[7],NULL);

    pthread_join(threadBF,NULL);

    printf("Real Time Processing completed ... Please close display window to exit .... \n");

    pthread_join(threadDraw,NULL);

    return 0;

    }
