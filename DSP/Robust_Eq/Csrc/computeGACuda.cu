#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
//#include <complex.h>

#include "mex.h"
#include "matrix.h"

#include "gpu/mxGPUArray.h"

/*
* mex -v -largeArrayDims computeGACuda.cu -lstdc++ -lc
*/

/*
 * Device code
 */
void __global__ doComputeGPU(double *GC,double *AReal,double *AImag,
							 double *hCapReal,double *hCapImag,double *curUReal,double *curUImag,
							 int N,double beta,double delNLMS,double eta,double hCapMod
							)
{
	int const i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N) 
	{
        GC[i] = ((1-beta)/(2*N))+(((1+beta)*(hCapReal[i]+hCapImag[i]))/(2*hCapMod+eta));
		AReal[i] = GC[i]*curUReal[i];
		AImag[i] = GC[i]*curUImag[i];
    }
} 

/*
	[GC,AReal,AImag,del] = computeGC(hCapReal,hCapImag,curUReal,curUImag,gDim,beta,delNLMS,eta);	
*/
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
	double hCapMod = 0;
	int i,N;

    int gDim;
	double beta;
	double delNLMS;
	double eta;
	double del;

	int const threadsPerBlock = 256;
    int blocksPerGrid;

    mxGPUArray const *hCapReal;
    mxGPUArray const *hCapImag;
    mxGPUArray const *curUReal;
    mxGPUArray const *curUImag;
    mxGPUArray *GC;
    mxGPUArray *AReal;
    mxGPUArray *AImag;

    double *d_hCapReal;
    double *d_hCapImag;
    double *d_curUReal;
    double *d_curUImag;
    double *d_GC;
    double *d_AReal;
    double *d_AImag;

    gDim = (int)mxGetScalar(prhs[4]);
	beta = (double)mxGetScalar(prhs[5]);
	delNLMS = (double)mxGetScalar(prhs[6]);
	eta = (double)mxGetScalar(prhs[7]);
	hCapMod = (double)mxGetScalar(prhs[8]);

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();

	hCapReal = mxGPUCreateFromMxArray(prhs[0]);
	hCapImag = mxGPUCreateFromMxArray(prhs[1]);
	curUReal = mxGPUCreateFromMxArray(prhs[2]);
	curUImag = mxGPUCreateFromMxArray(prhs[3]);

	d_hCapReal = (double *)(mxGPUGetDataReadOnly(hCapReal));
	d_hCapImag = (double *)(mxGPUGetDataReadOnly(hCapImag));
	d_curUReal = (double *)(mxGPUGetDataReadOnly(curUReal));
	d_curUImag = (double *)(mxGPUGetDataReadOnly(curUImag));

	GC = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(hCapReal),
                            mxGPUGetDimensions(hCapReal),
                            mxGPUGetClassID(hCapReal),
                            mxREAL,
                            MX_GPU_DO_NOT_INITIALIZE);
    d_GC = (double *)(mxGPUGetData(GC));

    AReal = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(hCapReal),
                            mxGPUGetDimensions(hCapReal),
                            mxGPUGetClassID(hCapReal),
                            mxREAL,
                            MX_GPU_DO_NOT_INITIALIZE);
    d_AReal = (double *)(mxGPUGetData(AReal));

    AImag = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(hCapReal),
                            mxGPUGetDimensions(hCapReal),
                            mxGPUGetClassID(hCapReal),
                            mxREAL,
                            MX_GPU_DO_NOT_INITIALIZE);
    d_AImag = (double *)(mxGPUGetData(AImag));

    N = (int)(mxGPUGetNumberOfElements(hCapReal));
    blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    /*  Kernel goes here */
    doComputeGPU<<<blocksPerGrid, threadsPerBlock>>>(d_GC,d_AReal,d_AImag,
							 					 d_hCapReal,d_hCapImag,d_curUReal,d_curUImag,
							 					 N,beta,delNLMS,eta,hCapMod
												);

    /* Wrap the result up as a MATLAB gpuArray for return. */
    plhs[0] = mxGPUCreateMxArrayOnGPU(GC);
    plhs[1] = mxGPUCreateMxArrayOnGPU(AReal);
    plhs[2] = mxGPUCreateMxArrayOnGPU(AImag);

    del = ((1-beta)/(2*gDim))*delNLMS;
    plhs[3] = mxCreateDoubleScalar(del);
    /*
     * The mxGPUArray pointers are host-side structures that refer to device
     * data. These must be destroyed before leaving the MEX function.
     */
    mxGPUDestroyGPUArray(hCapReal);
    mxGPUDestroyGPUArray(hCapImag);
    mxGPUDestroyGPUArray(curUReal);
    mxGPUDestroyGPUArray(curUImag);
    mxGPUDestroyGPUArray(GC);
    mxGPUDestroyGPUArray(AReal);
    mxGPUDestroyGPUArray(AImag);
}