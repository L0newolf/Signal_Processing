#include "hip/hip_runtime.h"
/*##################################################################################################*/
/*##################################################################################################

THIS IS THE AIO REAL-TIME VERSION.

/*##################################################################################################*/
/*##################################################################################################*/

// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <string.h>
#include <sys/types.h>
#include <sys/socket.h>
#include <sys/un.h>
#include <unistd.h>
#include <pthread.h>
#include <math.h>

// includes, cuda
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions
// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <hip/hip_vector_types.h>

#include "server_app.h"
using namespace std;

#define MAX2(x,y)      ((x)<(y) ? (y) : (x))

#define NUM_DAQS 8

//Macros for CUDA 5.5 compatibility
#define CUT_SAFE_CALL(x) checkCudaErrors((hipError_t)x)
#define cutGetMaxGflopsDeviceId() gpuGetMaxGflopsDeviceId()

StopWatchInterface *timer = NULL;

float hDFTCos[FBINS * DFTSIZE];
float hDFTSin[FBINS * DFTSIZE];
float hScale[SENSORS];

__constant__ float dDFTCos[FBINS * DFTSIZE];
__constant__ float dDFTSin[FBINS * DFTSIZE];
__constant__ float dScale[SENSORS];

float hBFRe[FBINS * BEAMS * SENSORS];
float hBFIm[FBINS * BEAMS * SENSORS];

float* dBFRe;
float* dBFIm;

short *hSensorData;
float hBFData[2*FBINS * BEAMS * BLOCKS];
float hostBFO[FBINS * BEAMS * BLOCKS];

short* dSensorData;
float* dBFData;

//Struct for pre-recorded beam processing thread parameters
struct beamRecordedArgs {
    char *dataPath;
    char *bfOutputFile;
    };

float hbfoData [FRAMES_PER_BLOCK*BEAMS*FBINS];
float *dbfoData;
float *curDispBuf;
float *devBFO;
float *devBFOCopy;
int dispBufPtr;

int bfDataCount = 0, totalBF = 0;

//Buffer pointer to store the display-processed beam frames
float frameBuffer[NUM_SQRS];

// Common variables used for data processing and display
int beams,curFrame,dataIdx,fRange,bfoBufPtr,drawBufPtr;
int frameCount = 0;
float cimg[ybeams][xbeams];

float dispBufBank[DISPLAY_BUFS][FRAMES_PER_BLOCK*BEAMS*FBINS];
int disBufStatus[DISPLAY_BUFS];
int disBufFrames[DISPLAY_BUFS];


short daqBuf[FILES][BUFFER_SIZE/sizeof(short)];

/*************************************/

//################################################################################################//
/*------------------------------------------------------------------------------------------------*/
/*------------------------------------------------------------------------------------------------*/
/*------------------------------------------------------------------------------------------------*/

/* initCuda() : Select the GPU to be used and initialize it  */
void initCuda(void) {

    CUT_SAFE_CALL(hipDeviceReset());
    int dev = cutGetMaxGflopsDeviceId();
    printf("MAX Device ID : %d\n", dev);
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, dev);
    printf("DEIVE NAME : %s\n", properties.name);
    CUT_SAFE_CALL(hipSetDevice(dev));
    CUT_SAFE_CALL(gpuDeviceInit(dev));
    }

/* checkCudaMem() : Check for available memory in the GPU   */
void checkCudaMem(void) {
    size_t total, avail;
    CUT_SAFE_CALL(hipMemGetInfo(&avail, &total));
    printf("CUDA Memory: total = %ld, free = %ld\n", total, avail);
    }

/* allocCudaMem() : Allocate memory in the GPU */
void allocCudaMem(void) {
    CUT_SAFE_CALL(hipMalloc((void**)&dSensorData, sizeof(short)*SENSORS*DFTSIZE*BLOCKS));
    CUT_SAFE_CALL(hipMalloc((void** )&dBFData, sizeof(hBFData)));
    CUT_SAFE_CALL(hipMalloc((void** )&dBFRe, sizeof(hBFRe)));
    CUT_SAFE_CALL(hipMalloc((void** )&dBFIm, sizeof(hBFIm)));
    CUT_SAFE_CALL(hipMalloc((void** )&devBFO, sizeof(hostBFO)));
    CUT_SAFE_CALL(hipMalloc((void** )&devBFOCopy, sizeof(hostBFO)));
    CUT_SAFE_CALL(hipMalloc((void** )&dbfoData, sizeof(hbfoData)));
    }

/* shutdown() : Free up GPU memory and reset GPU  */
void shutdown(void) {
    CUT_SAFE_CALL(hipFree(dBFRe));
    CUT_SAFE_CALL(hipFree(dBFIm));
    CUT_SAFE_CALL(hipFree(dBFData));
    CUT_SAFE_CALL(hipFree(dSensorData));
    CUT_SAFE_CALL(hipFree(devBFO));
    CUT_SAFE_CALL(hipFree(dbfoData));
    CUT_SAFE_CALL(hipDeviceReset());
    }

/* initDFT() : Initialize the DFT arrays  */
void initDFT(void) {
    float K = 2 * PI / DFTSIZE;
    for (int i = 0; i < DFTSIZE; i++) {
        for (int f = 0; f < FBINS; f++) {
            hDFTCos[f * DFTSIZE + i] = cos(K * (f + FBIN0) * i);
            hDFTSin[f * DFTSIZE + i] = -sin(K * (f + FBIN0) * i);
            }
        }
    CUT_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dDFTSin), hDFTSin, sizeof(hDFTSin)));
    CUT_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dDFTCos), hDFTCos, sizeof(hDFTCos)));
    }

/* initBF() : Initialize the beamformer arrays  */
void initBF() {
    for (int e = 0; e < EBEAMS; e++) {
        float phi = e * BEAMSPC - (EBEAMS - 1) * BEAMSPC / 2;
        for (int a = 0; a < ABEAMS; a++) {
            float theta = a * BEAMSPC - (ABEAMS - 1) * BEAMSPC / 2;
            for (int f = 0; f < FBINS; f++) {
                int fb = FBIN0 + f;
                for (int i = 0; i < SENSORS; i++) {
                    float w = sensorXPos[i] * sin(theta) * cos(phi)
                              + sensorYPos[i] * sin(phi);
                    w *= 2 * PI * RATE / DFTSIZE * fb / SOUNDSPEED;
                    int tn = (f * SENSORS + i) * BEAMS + e * ABEAMS + a;
                    hBFRe[tn] = sin(w) * hScale[i];
                    hBFIm[tn] = cos(w) * hScale[i];
                    }
                }
            }
        }
    CUT_SAFE_CALL(hipMemcpy(dBFRe, hBFRe, sizeof(hBFRe), hipMemcpyHostToDevice));
    CUT_SAFE_CALL(hipMemcpy(dBFIm, hBFIm, sizeof(hBFIm), hipMemcpyHostToDevice));
    }

/* readScale() : Read calibration of sensors  */
void readScale() {

    FILE* fh = fopen("calib.txt", "rt");
    if (fh == NULL) {
        printf("Sensor calibration not available\n");
        for (int i = 0; i < SENSORS; i++)
            hScale[i] = 1;
        }
    else {
        for (int i = 0; i < SENSORS; i++) {
            float s, o;
            fscanf(fh, "%f %f", &s, &o);
            hScale[i] = s;
            }
        fclose(fh);
        }
    CUT_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dScale), hScale, sizeof(hScale)));
    }


/* process() : beam-former CUDA kernel  */
__global__ void process(short* dataIn, float* dataOut,float* dBFRe,float* dBFIm, float *devBFO) {
    __shared__ float re[SENSORS];
    __shared__ float im[SENSORS];


    int f = blockIdx.x;
    int n = blockIdx.y;
    int s = threadIdx.x;
    if (s < SENSORS) {
        float sumRe = 0;
        float sumIm = 0;
        for (int i = 0; i < DFTSIZE; i++) {
            sumRe += dDFTCos[f * DFTSIZE + i]* dataIn[(n * DFTSIZE + i) * SENSORS + s];
            sumIm += dDFTSin[f * DFTSIZE + i]* dataIn[(n * DFTSIZE + i) * SENSORS + s];
            }
        re[s] = sumRe;
        im[s] = sumIm;
        }
    __syncthreads();
    if (s < BEAMBLKS * BEAMS) {
        int b = s % BEAMS;
        int start = s / BEAMS * SENSORS / BEAMBLKS;
        float sumRe = 0;
        float sumIm = 0;
        for (int i = start; i < start + SENSORS / BEAMBLKS; i++) {
            int tn = (f * SENSORS + i) * BEAMS + b;
            float tr = re[i] * dBFRe[tn] - im[i] * dBFIm[tn];
            float ti = im[i] * dBFRe[tn] + re[i] * dBFIm[tn];
            sumRe += tr;
            sumIm += ti;
            }

        int outndx2 = 2 * ((n * BEAMS + b) * FBINS + f);
        if (s < BEAMS) {
            dataOut[outndx2] = sumRe;
            dataOut[outndx2 + 1] = sumIm;
            }

        __syncthreads();
        if (s >= BEAMS) {
            dataOut[outndx2] += sumRe;
            dataOut[outndx2 + 1] += sumIm;
            }

        }

    __syncthreads();
     int b = s % BEAMS;
    int outndx2 = 2 * ((n * BEAMS + b) * FBINS + f);
    //devBFO[outndx2/2] = sqrt(pow(dataOut[outndx2],2)+pow(dataOut[outndx2 + 1],2));
    devBFO[outndx2/2] = (fabs(dataOut[outndx2])+fabs(dataOut[outndx2 + 1]))/2;
    }

/* ###################################################################################*/
/* ###################################################################################*/
template <typename T>
std::string to_string(T value) {
    std::ostringstream os ;
    os << value ;
    return os.str() ;
    }


// Initializes all the parameters and variables
void initVars() {
    fRange = freqBinUpper-freqBinLower+1;
    beams = ybeams*xbeams*fbins;
    curFrame = 0;
    //intialize the cimg array to all zeros
    for(int y=0; y<ybeams; y++) {
        for(int x=0; x<xbeams; x++) {
            cimg[y][x]=0.0f;
            }
        }
    dataIdx = 0;

    for(int i=0; i<DISPLAY_BUFS; i++) {
        for(int j=0; j<FRAMES_PER_BLOCK*BEAMS*FBINS; j++) {
            dispBufBank[i][j] = 0.0;
            }
        disBufStatus[i] = HOLD;
        }

    dispBufPtr = 0;
    }
/* ###################################################################################*/
/* ###################################################################################*/
// Interpolation Function
float interpolate (float in[ybeams][xbeams], float out[ybeamsInt][xbeamsInt]) {

    float xStep,yStep;
    float maxVal = 0.0;
    /* Fill all first rows */
    for(int j=0; j<ybeams; j++) {
        for(int i = 0; i<xbeams-1; i++) {
            xStep = (in[j][i+1] - in[j][i])/(INTERPOLATION_FACTOR);
            out[INTERPOLATION_FACTOR*j][INTERPOLATION_FACTOR*i] = in[j][i];
            for(int k=1; k<INTERPOLATION_FACTOR; k++) {
                out[INTERPOLATION_FACTOR*j][INTERPOLATION_FACTOR*i+k] = in[j][i] + k*xStep;
                if (maxVal<out[INTERPOLATION_FACTOR*j][INTERPOLATION_FACTOR*i+k])
                    maxVal = out[INTERPOLATION_FACTOR*j][INTERPOLATION_FACTOR*i+k];
                }
            }
        out[INTERPOLATION_FACTOR*j][xbeamsInt-1] = in[j][xbeams-1];
        if (maxVal<out[INTERPOLATION_FACTOR*j][xbeamsInt-1])
            maxVal = out[INTERPOLATION_FACTOR*j][xbeamsInt-1];
        }

    /* Fill up all the clomuns */
    for(int i=0; i<ybeams-1; i++) {
        for(int j=0; j<xbeamsInt; j++) {
            yStep = (out[INTERPOLATION_FACTOR*(i+1)][j] - out[INTERPOLATION_FACTOR*i][j])/INTERPOLATION_FACTOR;
            for(int k=1; k<INTERPOLATION_FACTOR; k++) {
                out[INTERPOLATION_FACTOR*i+k][j] = out[INTERPOLATION_FACTOR*i][j] + k*yStep;
                if (maxVal<out[INTERPOLATION_FACTOR*i+k][j])
                    maxVal = out[INTERPOLATION_FACTOR*i+k][j];

                }
            }
        }

    return 20*log10(maxVal);
    }
/* ###################################################################################
                    Beamformer Output Processing
   ################################################################################### */

void preprocBF(float *hBFData,float *bfoData, int framesToDraw ) {

    float bfoSum = 0.0;

    int k;

    for(k=0; k<framesToDraw; k++) {

        int i,j;
        for( i=0; i<beams; i++) {
            bfoSum = 0.0;
            for( j=0; j<FrameSize; j++) {
                bfoSum += hBFData[FrameSize*beams*k+beams*j+i];
                }
            bfoData[beams*k+i] = bfoSum/FrameSize;
            }

        }
    }


__global__ void preProcBF(float *dataIn, float *dataOut) {

    int dataPtr = threadIdx.x * FBINS*BEAMS*FRAMESIZE;
    int dataOutPtr = threadIdx.x * FBINS*BEAMS;
    float *threadBuf = dataIn + dataPtr;
    int i,j,dataCount = 0;;
    float bfoSum;

    for( i=0; i<FBINS*BEAMS; i++) {
            bfoSum = 0.0;
            dataCount = i;
            for( j=0; j<FRAMESIZE; j++) {
                bfoSum += threadBuf[dataCount];
                dataCount+=BEAMS*FBINS;
                }
            dataOut[dataOutPtr+i] = bfoSum/FRAMESIZE;
            }

    }


void processBF(float *bfOutput) {

    float sum,maxVal;
    float beamFrames[xbeams][ybeams][fbins];
    float frameFreqAvg[xbeams][ybeams];
    float cimgInt[ybeamsInt][xbeamsInt];
    int bufIdx = 0;
    int dynAdd;

    sum =0;
    maxVal = 0;

//Select the frames in blocks of Framesize,Average out the frames over the desired frequency range

    for(int y=ybeams-1; y>-1; y--) {
        for(int x=xbeams-1; x>-1; x--) {

            dataIdx += FBINS*BEAMS*(freqBinLower-1);
            for(int f=0; f<fbins; f++) {
                beamFrames[x][y][f] = bfOutput[dataIdx];
                dataIdx++;
                }
            sum = 0.0f;
            for(int f=freqBinLower-1; f<freqBinUpper; f++) {
                sum += beamFrames[x][y][f];
                }
            frameFreqAvg[x][y] = sum/fRange;
            cimg[y][x] = ExpAvg*cimg[y][x] + (1.0-ExpAvg)*(sum/fRange);
            }
        }


//Interpolate cimg
    maxVal = interpolate(cimg,cimgInt);
    dynAdd = -maxVal+DynRange;

//Take log of interpolated cimg and do dynamic ranging
float temp;
    for(int y=0; y<ybeamsInt; y++) {
        for(int x=0; x<xbeamsInt; x++) {
            temp = (20*log10(cimgInt[y][x])+dynAdd)/DynRange;
            if(temp<0)
                frameBuffer[y*xbeamsInt + x +y-2] = 0.0;
            else if(temp>1.0)
                frameBuffer[y*xbeamsInt + x +y-2] = 63.0;
            else
                frameBuffer[y*xbeamsInt + x +y-2] = 63.0*temp;
            }
        }
    }


/******************BEAM PROCESSING FOR PRE-RECORDED DATA SET********************************************/

void beamProcess(char *directory) {

    printf("Starting beam-forming .... \n");

    int i=0,blks;
    int samples = 0;

    int sleepCount = 0;

    readScale();

    initCuda();
    checkCudaMem();
    initSensors();
    allocCudaMem();
    checkCudaMem();
    initDFT();
    initBF();

    int offSet = 0;
    int dataSize = 0 ;
    bfDataCount = 0;
    totalBF = 0;
    printf("Beam-former Initialized ... Start DAQ accquisition ... \n");

    hSensorData = (short *)malloc(SENSORS*SAMPLES*sizeof(short));

    FILE* fh[FILES];
    for (int a = 0; a < FILES; a++) {
    char fn[256];
    sprintf(fn, "%s/DAQ_%d",directory, a + 1);

    fh[a] = fopen(fn, "rb");
    if(fh[a]==NULL)
        printf("File open failed for DAQ_%d... \n",a );
    fseek(fh[a], 0, SEEK_END);
		int n = ftell(fh[a]);
		if (a == 0)
			{dataSize = n;}
		else if (n < dataSize)
			{dataSize = n;}
	fseek(fh[a], 0, SEEK_SET);
    }

    printf("dataSize : %d buffer size : %d\n",dataSize,BUFFER_SIZE);
    int loops = dataSize/(8*1024*1024);
    printf("loops : %d\n",loops);
	printf("All files opened... \n");

    while(totalBF<loops){

    for (int a = 0; a < FILES; a++) {
        fread(&daqBuf[a][0], BUFFER_SIZE, 1, fh[a]);
        }

    samples = (BUFFER_SIZE- offSet)/( (64 * sizeof(short) + 4));

    int daqPtr = offSet/2 ,bufPtr = 0;

    short *curBuf;

    for (int j = 0; j < samples; j++) {

                daqPtr+=2;

                curBuf = &daqBuf[0][0] ;
                memcpy(hSensorData+bufPtr,curBuf+daqPtr,64*sizeof(short));
                bufPtr+=64;

                curBuf = &daqBuf[1][0] ;
                memcpy(hSensorData+bufPtr,curBuf+daqPtr,64*sizeof(short));
                bufPtr+=64;

                curBuf = &daqBuf[2][0] ;
                memcpy(hSensorData+bufPtr,curBuf+daqPtr,64*sizeof(short));
                bufPtr+=64;

                curBuf = &daqBuf[3][0] ;
                memcpy(hSensorData+bufPtr,curBuf+daqPtr,64*sizeof(short));
                bufPtr+=64;

                curBuf = &daqBuf[4][0] ;
                memcpy(hSensorData+bufPtr,curBuf+daqPtr,64*sizeof(short));
                bufPtr+=64;

                curBuf = &daqBuf[5][0] ;
                memcpy(hSensorData+bufPtr,curBuf+daqPtr,64*sizeof(short));
                bufPtr+=64;

                curBuf = &daqBuf[6][0] ;
                memcpy(hSensorData+bufPtr,curBuf+daqPtr,64*sizeof(short));
                bufPtr+=64;

                curBuf = &daqBuf[7][0] ;
                memcpy(hSensorData+bufPtr,curBuf+daqPtr,60*sizeof(short));
                bufPtr+=60;

                daqPtr+=64;


    }

    offSet = 132 - (BUFFER_SIZE - offSet) % (64 * sizeof(short) + 4);

    i=0;

    while (i < SAMPLES) {
        blks = (SAMPLES - i) / DFTSIZE;
        if (blks > BLOCKS)
            blks = BLOCKS;


        CUT_SAFE_CALL(hipMemcpy(dSensorData, hSensorData+SENSORS*i, DFTSIZE*SENSORS*blks*sizeof(short), hipMemcpyHostToDevice));
        dim3 threads(MAX2(SENSORS,BEAMS*BEAMBLKS));
        dim3 grid(FBINS, blks);
        process<<<grid, threads>>>(dSensorData, dBFData, dBFRe, dBFIm,devBFO);

        dim3 threads1(blks/FRAMESIZE);
        dim3 grid1(1, 1);
        preProcBF<<<grid1, threads1>>>(devBFO, dbfoData);
        CUT_SAFE_CALL(hipMemcpy(hbfoData, dbfoData, sizeof(hbfoData),hipMemcpyDeviceToHost));


        for(int j=0; j<(blks/FRAMESIZE)*BEAMS*FBINS; j++){
            dispBufBank[bfDataCount%DISPLAY_BUFS][j] = hbfoData[j];
            }
        disBufFrames[bfDataCount%DISPLAY_BUFS] = blks/FRAMESIZE;
        disBufStatus[bfDataCount%DISPLAY_BUFS] = DRAW;
        bfDataCount++;


        i += DFTSIZE * BLOCKS;



        }



    totalBF++;
}

    shutdown();
    printf("Total data blocks processed in BF : %d\n",totalBF);
    printf("Total blocks sent to display : %d\n",bfDataCount);
    printf("Sleep count : %d\n",sleepCount);
    pthread_exit(0);
    }
/**********************************************************************************************/

/* ###################################################################################
					Beamformer Display using OpenGL
   ################################################################################### */

void printText(int x, int y, string textDisplay) {

    glMatrixMode(GL_PROJECTION);
    glPushMatrix();
    glLoadIdentity();
    glOrtho(0, WIN_WIDTH, 0, WIN_HEIGHT, -1.0f, 1.0f);
    glMatrixMode(GL_MODELVIEW);
    glPushMatrix();
    glLoadIdentity();
    glPushAttrib(GL_DEPTH_TEST);
    glDisable(GL_DEPTH_TEST);
    glRasterPos2i(x,y);
    for (int i=0; i<textDisplay.size(); i++) {
        glutBitmapCharacter(GLUT_BITMAP_HELVETICA_12, textDisplay[i]);
        }
    glPopAttrib();
    glMatrixMode(GL_PROJECTION);
    glPopMatrix();
    glMatrixMode(GL_MODELVIEW);
    glPopMatrix();
    }

void displayMsg() {

    //Set Colour to white
    glColor3f(1.0,1.0,1.0);
    string textDisplay;

    //Display all option parameters
    textDisplay="Current Lower Frequency Bin : "+to_string(freqBinLower);
    printText(10,7,textDisplay);
    textDisplay="Current Lower Frequency Bin : "+to_string(freqBinUpper);
    printText(10,24,textDisplay);
    textDisplay="Current Dynamic Range Limit : "+to_string(DynRange);
    printText(10,41,textDisplay);

    //Display guide to change values of the parameters
    textDisplay="Press UP key to raise and  DOWN key to lower the Lower Frequency Bin limit";
    printText(270,41,textDisplay);
    textDisplay="Press RIGHT key to raise and LEFT key to lower the Upper Frequency Bin limit";
    printText(270,24,textDisplay);

    //Display Cloour bar text
    textDisplay="Colour Bar for Signal Strength  : ";
    printText(10,115,textDisplay);

    //Display color bar values
    textDisplay="  0               10               20               30               40                50               60               70                 80 ";
    printText(30,80,textDisplay);

    //Display azimuthal angle values
    textDisplay="-8.8 -7.2  -6.4  -5.6  -4.8  -4.0  -3.2  -2.4  -1.6  -0.8   0.0   0.8   1.6   2.4   3.2    4.0   4.8   5.6   6.4   7.2   8.0   8.8";
    printText(25,165,textDisplay);
    textDisplay="Azimuthal";
    printText(320,150,textDisplay);

    //Display Elevation angle values
    textDisplay="-4.0";
    printText(10,180,textDisplay);
    textDisplay="-3.2";
    printText(10,213.5,textDisplay);
    textDisplay="-2.4";
    printText(10,247,textDisplay);
    textDisplay="-1.6";
    printText(10,280.5,textDisplay);
    textDisplay="-0.8";
    printText(10,314,textDisplay);
    textDisplay="0.0";
    printText(15,347.5,textDisplay);
    textDisplay="0.8";
    printText(15,381,textDisplay);
    textDisplay="1.6";
    printText(15,414.5,textDisplay);
    textDisplay="2.4";
    printText(15,448,textDisplay);
    textDisplay="3.2";
    printText(15,481.5,textDisplay);
    textDisplay="4.0";
    printText(15,515,textDisplay);

    //Display "Elevation" as text
    textDisplay="E";
    printText(700,420,textDisplay);
    textDisplay="l";
    printText(700,405,textDisplay);
    textDisplay="e";
    printText(700,390,textDisplay);
    textDisplay="v";
    printText(700,375,textDisplay);
    textDisplay="a";
    printText(700,360,textDisplay);
    textDisplay="t";
    printText(700,345,textDisplay);
    textDisplay="i";
    printText(700,330,textDisplay);
    textDisplay="o";
    printText(700,315,textDisplay);
    textDisplay="n";
    printText(700,300,textDisplay);


    }
//! Create a timer event for refresh
void timerEvent(int value) {
    if(disBufStatus[dispBufPtr] == DRAW) {
        updateFrame();
        glutPostRedisplay();
        }
    glutTimerFunc(refreshDelay, timerEvent, 0);
    }

void genVerticesDisplay() {

    int i;
    int x=0;
    int y=0;
    int clr;
    float xcords[xPoints];
    float ycords[yPoints];

    int clrPtr;
    float xstep = (2*xLim)/(float)(xPoints-1);
    float ystep = (2*yLim)/(float)(yPoints-1);

    for(i=0; i<xPoints; i++)
        xcords[i] = -xLim + i*xstep;

    for(i=0; i<yPoints; i++)
        ycords[i] = -yLim + i*ystep+yOffset;

    clrPtr = 0;

    for (i=0; i<NUM_SQRS; i++) {
        if (x==xPoints-1) {
            x=0;
            y=y+1;
            }

        clr = (int)(frameBuffer[clrPtr]/2.0);
        if (clr < MAX_COLOURS)
            glColor3f(accousColor[clr][0],accousColor[clr][1],accousColor[clr][2]);
        else
            glColor3f(accousColor[MAX_COLOURS-1][0],accousColor[MAX_COLOURS-1][1],accousColor[MAX_COLOURS-1][2]);

        glVertex3f(xcords[x], ycords[y], 0.0);
        glVertex3f(xcords[x+1], ycords[y], 0.0);
        glVertex3f(xcords[x+1],  ycords[y+1], 0.0);
        glVertex3f( xcords[x], ycords[y+1], 0.0);

        x+=1;
        clrPtr+=1;

        }

    //Draw the colour bar
    x=0;
    float yColorBar[] = {-0.65,-0.6};
    int curColour = 0;
    int drawCnt = 0;

    for(i=0; i<4*xPoints; i+=4) {

        glColor3f(accousColor[curColour][0],accousColor[curColour][1],accousColor[curColour][2]);
        glVertex3f(xcords[x], yColorBar[0], 0.0);
        glVertex3f(xcords[x+1], yColorBar[0], 0.0);
        glVertex3f(xcords[x+1],  yColorBar[1], 0.0);
        glVertex3f( xcords[x], yColorBar[1], 0.0);

        drawCnt+=1;
        if(drawCnt % 5 == 0)
            curColour++;

        x+=1;
        }


    }

void fpsDisplay() {
    char fps[64];
    curTime = (float)frameCount*timePerFrame;
    sprintf(fps, "ROMANIS RT DISPLAY  Current Time : %0.3f  Frame Number : %d",curTime,frameCount);
    glutSetWindowTitle(fps);
    }

void display(void) {

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glEnable(GL_DEPTH_TEST);
    glClear( GL_COLOR_BUFFER_BIT);

    glShadeModel(GL_SMOOTH);
    glBegin(GL_QUADS);

    genVerticesDisplay();

    glEnd();
    glFlush();

    displayMsg();

    fpsDisplay();

    frameCount++;
    curFrame++;
    glutSwapBuffers();



    }

void special(int key, int x, int y) {
    switch (key) {
        case GLUT_KEY_LEFT:
            if(freqBinUpper > freqBinLower)
                freqBinUpper--;
            break;
        case GLUT_KEY_RIGHT:
            if(freqBinUpper<FBINS-1)
                freqBinUpper++;
            break;
        case GLUT_KEY_UP:
            if(freqBinLower<freqBinUpper)
                freqBinLower++;
            break;
        case GLUT_KEY_DOWN:
            if(freqBinLower>0)
                freqBinLower--;
            break;
        }

    glutPostRedisplay();
    }

void drawFrame(int argc, char *argv[]) {

    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
    glutInitWindowSize(WIN_WIDTH, WIN_HEIGHT);
    glutInitWindowPosition(0, 0);

    glutCreateWindow("ROMANIS BEAM VISULISATION ");

    glutTimerFunc(refreshDelay, timerEvent, 0);
    glutDisplayFunc(display);
    glutSpecialFunc(special);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glutMainLoop();
    }


void * startDraw(void *args) {

    char fakeParam[] = "fake";
    char *fakeargv[] = { fakeParam, NULL };
    int fakeargc = 1;
    initVars();
    drawFrame(fakeargc,fakeargv);

    pthread_exit(0);
    }

void updateFrame() {


    if(curFrame == disBufFrames[dispBufPtr] && frameCount!=0) {
        dataIdx = 0;
        curFrame = 0;
        memset(&dispBufBank[dispBufPtr][0],0.0,(FRAMES_PER_BLOCK*BEAMS*FBINS)*sizeof(float));
        disBufStatus[dispBufPtr] = HOLD;
        dispBufPtr++;
        if(dispBufPtr == DISPLAY_BUFS ) dispBufPtr = 0;

        }

    curDispBuf = &dispBufBank[dispBufPtr][0];

    processBF(curDispBuf);

    }
/* ###################################################################################
					 Main function
   ################################################################################### */
int main(int argc, char* argv[]) {

    if (argc < 2) {
		printf(
				"Usage: %s Path_to_DAQ_dataset output_file \n",argv[0]);

		return 1;
	}

	char *directory = argv[1];
    initVars();

    printf("Launching Draw thread now ... \n");

    pthread_t threadDraw;
    void *dummy = 0x0000;
    pthread_create(&threadDraw,NULL,startDraw,dummy);

    printf("Starting beam-former function .... \n");
    beamProcess(directory);

    printf("Real Time Processing completed ... Please close display window to exit .... \n");

    pthread_join(threadDraw,NULL);

    return 0;

    }
